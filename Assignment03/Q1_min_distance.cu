#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <sys/time.h>


// kernel config cuda
#define BLOCK_SIZE 256


// Funtion to get the current time in seconds.
double current_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}


// Implementation of CPU Serial (Part 1)
// The method find the minimum Euclidean distance between 
// unique pairs of particles in 2D space using a serial implementation on cpu
float cpu_serial_min_distance(const float *x, const float *y, int n) {
    float dist_minimum = FLT_MAX;
    // we iterates through unique particle pairs , j > i only
    for (int i = 0; i < n; i++) {
        for (int j = i + 1; j < n; j++) { 
            // Euclidean distance part of formula (x2 - x1)
            float dx = x[i] - x[j];
            // Euclidean distance part of formula (y2 - y1)
            float dy = y[i] - y[j];
            // Final part of Euclidean distance sqrt(dx**2 + dy**2)
            float pair_dist = sqrtf(dx * dx + dy * dy);

            // Update the minimum distance
            if (pair_dist < dist_minimum) {
                dist_minimum = pair_dist;
            }
        }
    }
    return dist_minimum;
}


// Implementation of GPU One Thread per Particle (Part 2)
// In the funton each thread is assigned particle and computes the minimum distance.
// So each thread calculate the Euclidean distance between its particle and all other particle (not itself)
// And it stores the minimum Euclidean distance it finds in local array.
__global__
void min_distance_perParticle_gpu(const float *x, const float *y, int n, float *temp_local_min) {
    // compute thread global index 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

     // Make sure the thread is within bounds
    if (i < n) {
        float x_index = x[i];
        float y_index = y[i];

        // Initialization to largest possible float value
        float minimum_val = FLT_MAX;

        for (int j = 0; j < n; j++) {
            // Don't calculate the distance with itself
            if (j == i) continue;
            // calculate the euclidean distance.
            float d_x = x_index - x[j];
            float d_y = y_index - y[j];
            float local_dist = sqrtf(d_x * d_x + d_y * d_y);
            // update is smaller
            if (local_dist < minimum_val)
                minimum_val = local_dist;
        }
        temp_local_min[i] = minimum_val;
    }
}


// Implementation of GPU One Thread per Pair (Part 3)
// In this part in each thread it calculate the distance for one unique pair of particles.
// unique pairs total number ~= N*(N-1)/2.
__global__
void gpu_min_distance_per_pair(const float *x, const float *y, int n, float *pair_dist, long long total_Pairs) {
    // get the thread index (thread of unique pair)
    long long th_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // break if index of thread > unique pairs number
    if (th_idx >= total_Pairs) return;


    // This mapping walks through a conceptual upper-triangular matrix

    // Map the index to a unique pair (i, j) such that i < j.
    // in the mapping it walks thruugh an upper-triangular matrix to invert the index.
    int rowi = 0;
    long long tempopary = th_idx;


    // Find row (i) by doing subtraction of lengths of each row in the uper triangle until tempopary fits in
    while (tempopary >= (n - 1 - rowi)) {
        tempopary -= (n - 1 - rowi);
        rowi++;
    }

    // now find j, which is the offset in row i 
    int j = rowi + 1 + tempopary;

    float d_x = x[rowi] - x[j];
    float d_y = y[rowi] - y[j];
    float temppdist = sqrtf(d_x * d_x + d_y * d_y);
    pair_dist[th_idx] = temppdist;
}


// this part is kernel to do binary reduction and compute the minimum value in the input array.
// will use it in part 2 (GPU) and part 3 (GPU) for reduction.
__global__
void reduce_min_kernel(float *g_data, int n) {
    // define a memory shared to store minimum distances values (intermediate) in a block.
    extern __shared__ float shared_data[];
    // the thread index
    unsigned int thread_id = threadIdx.x;
    // index for the first  two elements in the thread
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // initiate to largest value
    float thread_Min = FLT_MAX;
     // put the first element into to thread min if in bounds.
    if (i < n) {
        thread_Min = g_data[i];
        // update thread_Min if smaller 
        if (i + blockDim.x < n) {
            float temp = g_data[i + blockDim.x];
            if (temp < thread_Min)
                thread_Min = temp;
        }
    }
    // save minimum value into shared memory to later reduction
    shared_data[thread_id] = thread_Min;

    // Synchronize the threads to make sure all at the same stage (all data is written)
    __syncthreads();

    // binary reduction on the shared memory
    for (unsigned int strid = blockDim.x / 2; strid > 0; strid >>= 1) {
        if (thread_id < strid) {
            float a = shared_data[thread_id];
            float b = shared_data[thread_id + strid];

            if (b < a)
            {
                shared_data[thread_id] = b;
            }
            else
            {
               shared_data[thread_id] = a; 
            }
           // shared_data[thread_id] = (b < a) ? b : a;
        }
        // Synchronize
        __syncthreads();
    }
    // save block result to global memory
    if (thread_id == 0) {
        g_data[blockIdx.x] = shared_data[0];
    }
}



int main(int argc, char **argv) {

    int N = 4096; 
    if (argc > 1) {
        N = atoi(argv[1]);
    }
    if (N <= 1) {
        fprintf(stderr, "(N): Must be greater than 1.\n");
        return 1;
    }

    // Use a fixed seed 
    srand(0);

    // Memory allocation for particle coordinates (cpu)
    float *memory_x = (float *)malloc(N * sizeof(float));
    float *memory_y = (float *)malloc(N * sizeof(float));

    // Random Number generation in range [0, 1].
    for (int ii = 0; ii < N; ii++) {
        memory_x[ii] = (float)rand() / (float)RAND_MAX;
        memory_y[ii] = (float)rand() / (float)RAND_MAX;
    }

    
    // Serial computation (Part 1)
    double start_time_epu = current_time();
    float min_euc_dis_cpu = cpu_serial_min_distance(memory_x, memory_y, N);
    double end_time_cpu = current_time();
    printf("Part 1:\nMinimum Euclidean Distance = %e, Time = %f seconds\n\n", min_euc_dis_cpu, end_time_cpu - start_time_epu);

    
    // Copy to GPU
    float *d_x, *d_y;
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));
    hipMemcpy(d_x, memory_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, memory_y, N * sizeof(float), hipMemcpyHostToDevice);

   

    // GPU one thread per particle (Part 2)
    // memory allocation for local minimum distances.
    float *min_dist_local;
    hipMalloc((void**)&min_dist_local, N * sizeof(float));

    int threadInBlock = BLOCK_SIZE;
    int PerGrid_blocks = (N + threadInBlock - 1) / threadInBlock;

    double start_time_gpu_p2 = current_time();
    min_distance_perParticle_gpu<<<PerGrid_blocks, threadInBlock>>>(d_x, d_y, N, min_dist_local);
    hipDeviceSynchronize();

    // reduce the local min_dist to a global.
    int Size_Cur = N;
    while (Size_Cur > 1) {
        int thrds = BLOCK_SIZE;
        int blocks = (Size_Cur + thrds * 2 - 1) / (thrds * 2);
        reduce_min_kernel<<<blocks, thrds, thrds * sizeof(float)>>>(min_dist_local, Size_Cur);
        hipDeviceSynchronize();
        Size_Cur = blocks;
    }
    float min_dist_gpu_p2;
    hipMemcpy(&min_dist_gpu_p2, min_dist_local, sizeof(float), hipMemcpyDeviceToHost);
    double end_time_gpu_p2 = current_time();

    printf("Part 2:\nMinimum Euclidean Distance = %e, Time = %f seconds\n\n", min_dist_gpu_p2, end_time_gpu_p2 - start_time_gpu_p2);

    // GPU with one thread per pair (Part 3)
    // unique pairs total
    long long total_Pairs = ((long long)N * (N - 1)) / 2;
    float *d_pair_dist;
    hipMalloc((void**)&d_pair_dist, total_Pairs * sizeof(float));

    int threadInBlockPair = BLOCK_SIZE;
    int blocksPairPerGrid = (total_Pairs + threadInBlockPair - 1) / threadInBlockPair;
    double start_time_gpu2 = current_time();
    gpu_min_distance_per_pair<<<blocksPairPerGrid, threadInBlockPair>>>(d_x, d_y, N, d_pair_dist, total_Pairs);
    hipDeviceSynchronize();

    // Reduce pair distances to get global minimum
    Size_Cur = total_Pairs;
    while (Size_Cur > 1) {
        int thrds = BLOCK_SIZE;
        int blocks = (Size_Cur + thrds * 2 - 1) / (thrds * 2);
        reduce_min_kernel<<<blocks, thrds, thrds * sizeof(float)>>>(d_pair_dist, Size_Cur);
        hipDeviceSynchronize();
        Size_Cur = blocks;
    }
    float min_euc_dist_gpu2;
    hipMemcpy(&min_euc_dist_gpu2, d_pair_dist, sizeof(float), hipMemcpyDeviceToHost);
    double end_time_gpu2 = current_time();
    printf("Part 3:\nMinimum Euclidean Distance = %e, Time = %f seconds\n\n", min_euc_dist_gpu2, end_time_gpu2 - start_time_gpu2);

    // Clean 
    hipFree(d_x);
    hipFree(d_y);
    hipFree(min_dist_local);
    hipFree(d_pair_dist);
    free(memory_x);
    free(memory_y);

    return 0;
}
